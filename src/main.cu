#include "hip/hip_runtime.h"
#include "rtweekend.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "hittable.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.cuh"
#include "clock.h"
#include "material.cuh"
#include <hiprand/hiprand_kernel.h>
#include "utility.cuh"
#include "material_manager.cuh"


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


__host__ void create_materials(material_manager* mat_manager) {
    printf("size of materials: \nlambertian: %lu\nmetal: %lu \ndieletric: %lu\n", sizeof(lambertian), sizeof(metal), sizeof(dieletric));
    material_info* ground = new material_info;
    ground->type = material_type::lambertian_t;
    ground->albedo = vec3(0.8, 0.8, 0.0);

    mat_manager->add_material(ground);

    material_info* mid = new material_info;
    mid->type = material_type::lambertian_t;
    mid->albedo = vec3(0.3, 0.3, 0.8);

    mat_manager->add_material(mid);

    material_info* right = new material_info;
    right->type   = material_type::metal_t;
    right->albedo = vec3(0.8, 0.6, 0.2);
    right->fuzz   = 0.2f;

    mat_manager->add_material(right);

    material_info* hollow_in = new material_info;
    hollow_in->type   = material_type::dieletric_t;
    hollow_in->refraction_index = 1.5f;

    mat_manager->add_material(hollow_in);

    material_info* hollow_out = new material_info;
    hollow_out->type   = material_type::dieletric_t;
    hollow_out->refraction_index = 1.0f / 1.5f;

    mat_manager->add_material(hollow_out);
}

// TODO: make create_materials again
__global__ void create_world(hittable **d_list, hittable **d_world, material_manager* mat_manager) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {

        material** mats = mat_manager->get_device_materials();

        d_list[0] = new sphere(vec3(0,0,-1.2f), 0.5,
                               mats[1]);
        d_list[1] = new sphere(vec3(0,-100.5,-1), 100,
                               mats[0]);
        d_list[2] = new sphere(vec3(1,0,-1), 0.5,
                               mats[2]);
        d_list[3] = new sphere(vec3(-1,0,-1), 0.5,
                               mats[3]);
        d_list[4] = new sphere(vec3(-1.0f, 0.0, -1.0), 0.4,
                               mats[4]);
        *d_world  = new hittable_list(d_list,5);
    }
}

__global__ void free_world(hittable **d_list, hittable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for(int i=0; i < 5; i++) {
            delete ((sphere *)d_list[i])->mat; // TODO: delete this stuff in mat_manager
            delete d_list[i];
        }
        delete *d_world;
    }
}

__global__ void render_init(int seed, int res_x, int res_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= res_x) || (j >= res_y)) return;
    int pixel_index = j*res_x + i;

    hiprand_init(seed, pixel_index, 0, &rand_state[pixel_index]);
}

// don't need to copy cam to device explicitly, it's copied by value by cuda automatically
// and doesn't impact much given that it's just one kernel call.
__global__ void render(camera cam, vec3* framebuffer, hittable** world, hiprandState *rand_state) {
    cam.render(framebuffer, world, rand_state);
}

void save_framebuffer_to_file(vec3* framebuffer, int res_x, int res_y, std::string output_filename) {
    std::ofstream file(output_filename);
    std::streambuf* coutBuffer = std::cout.rdbuf();
    std::cout.rdbuf(file.rdbuf());

    std::cout << "P3\n" << res_x << " " << res_y << "\n255\n";
    for (int j = 0; j < res_y; j++) {
        for (int i = 0; i < res_x; i++) {
            size_t pixel_index = j*res_x + i;
            vec3 rgb = framebuffer[pixel_index];

            write_color(std::cout, rgb);
        }
    }

    std::cout.rdbuf(coutBuffer);
    std::cout << "Finished" << std::endl;
}

camera create_camera() {
    camera cam;
    cam.aspect_ratio = 16.0f/9.0f;
    cam.res_x  = 400;
    cam.spp    = 100;
    cam.center = point3(-2, 2, 1);
    cam.lookat = point3(0, 0, -1);
    cam.up     = point3(0, 1, 0);
    cam.vfov   = 30;

    cam.defocus_angle  = 0.1f;
    cam.focal_distance = 10.0f;
    cam.initialize();
    return cam;
}

int main() {
    // Camera
    camera cam = create_camera();
    int res_y  = cam.get_res_y();

    // materials: 
    material_manager* mat_manager;
    checkCudaErrors(hipMallocManaged((void**)&mat_manager, sizeof(mat_manager)));
    create_materials(mat_manager);
    mat_manager->dispatch();

    create_device_materials<<<1, 1>>>(
        mat_manager->get_device_material_info(),
        mat_manager->get_device_materials(),
        mat_manager->size()
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Framebuffer
    int num_pixels = cam.res_x * res_y;
    size_t framebuffer_size = num_pixels * sizeof(vec3);
    vec3 *framebuffer;

    checkCudaErrors(hipMallocManaged((void **)&framebuffer, framebuffer_size));

    int threads_x = 8, threads_y = 8;
    dim3 threads(threads_x, threads_y);
    dim3 blocks(cam.res_x / threads_x  + 1, res_y / threads_y + 1);

    // world creation
    hittable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 5*sizeof(hittable *)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));

    create_world<<<1,1>>>(d_list,d_world, mat_manager);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hiprandState *rand_state;
    checkCudaErrors(hipMalloc((void **)&rand_state, num_pixels * sizeof(hiprandState)));

    render_init<<<blocks, threads>>>(1984, cam.res_x, res_y, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    rtweekend::clock c;

    c.start();
    render<<<blocks, threads>>>(cam, framebuffer, d_world, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    c.end();
    c.print();

    std::cout << "Saving framebuffer to file..." << std::endl;
    save_framebuffer_to_file(framebuffer, cam.res_x, res_y, "output/output.ppm");

    // freeing stuff
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(framebuffer));
    return 0;
}


