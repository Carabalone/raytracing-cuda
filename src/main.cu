#include "hip/hip_runtime.h"
#include "rtweekend.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "hittable.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.cuh"
#include "clock.h"
#include "material.cuh"
#include <hiprand/hiprand_kernel.h>
#include "color.h"
#include "material_manager.cuh"

__host__ void create_materials(material_manager* mat_manager) {
    printf("size of materials: \nlambertian: %lu\nmetal: %lu \ndieletric: %lu\n", sizeof(lambertian), sizeof(metal), sizeof(dieletric));

    // random 10 lambertian materials
    for (int i=0; i < 10; i++) {
        material_info* mat = new material_info;
        mat->type = material_type::lambertian_t;
        mat->albedo = random_color();
        mat_manager->add_material(mat);
    }

    // random 3 metal materials
    for (int i=0; i < 3; i++) {
        material_info* mat = new material_info;
        mat->type = material_type::metal_t;
        mat->albedo = random_color();
        mat->fuzz = float(random_double());
        mat_manager->add_material(mat);
    }

    // random 3 dieletric materials
    for (int i=0; i < 3; i++) {
        material_info* mat = new material_info;
        mat->type = material_type::dieletric_t;
        mat->refraction_index = float(random_double());
        mat_manager->add_material(mat);
    }

}

// TODO: make create_materials again
__global__ void create_world(hittable **d_list, hittable **d_world, material_manager* mat_manager, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {

        hiprandState local_rand_state = rand_state[0];
        material** mats = mat_manager->get_device_materials();

        // for (int a=-11; a < 11; a++) {
        //     for (int b=-11; b < 11; b++) {
        //         auto choose_mat_type = gpu_rand(local_rand_state);
        //         point3 center(a + 0.9 * gpu_rand(local_rand_state), 0.2, b + 0.9 * gpu_rand(local_rand_state));
        //
        //         if ((center - point3(4, 0.2, 0)).length() > 0.9f) {
        //             printf("acess: %d/%d\n", ((a+11) * (b+11) + (b+11)), (22*22));
        //             if (choose_mat_type < 0.8) {
        //                 auto choose_mat = int(gpu_rand(local_rand_state) * 10.5f);
        //                 printf("choose_mat: %d/%d\n", choose_mat, mat_manager->size());
        //                 d_list[(a + 11) * (b + 11) + b + 11] = new sphere(center, 0.2, mats[choose_mat]);
        //             } else if (choose_mat_type < 0.95) {
        //                 auto choose_mat = 11 + int(gpu_rand(local_rand_state) * 3);
        //                 printf("choose_mat: %d/%d\n", choose_mat, mat_manager->size());
        //                 d_list[(a + 11) * (b + 11) + b + 11] = new sphere(center, 0.2, mats[choose_mat]);
        //             } else {
        //                 auto choose_mat = 13 + int(gpu_rand(local_rand_state) * 3);
        //                 printf("choose_mat: %d/%d\n", choose_mat, mat_manager->size());
        //                 d_list[(a + 11) * (b + 11) + b + 11] = new sphere(center, 0.2, mats[choose_mat]);
        //             } 
        //         }
        //     }
        // }

        d_list[0] = new sphere(vec3(0,0,-1.2f), 0.5,
                               mats[1]);
        d_list[1] = new sphere(vec3(0,-100.5,-1), 100,
                               mats[0]);
        d_list[2] = new sphere(vec3(1,0,-1), 0.5,
                               mats[2]);
        d_list[3] = new sphere(vec3(-1,0,-1), 0.5,
                               mats[3]);
        d_list[4] = new sphere(vec3(-1.0f, 0.0, -1.0), 0.4,
                               mats[4]);
        *d_world  = new hittable_list(d_list, 5);
    }
}

__global__ void free_world(hittable **d_list, hittable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        // for(int i=0; i < 16; i++) {
        //     delete ((sphere *)d_list[i])->mat; // TODO: delete this stuff in mat_manager
        //     delete d_list[i];
        // }
        delete *d_world;
    }
}

__global__ void render_init(int seed, int res_x, int res_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= res_x) || (j >= res_y)) return;
    int pixel_index = j*res_x + i;

    hiprand_init(seed, pixel_index, 0, &rand_state[pixel_index]);
}

// don't need to copy cam to device explicitly, it's copied by value by cuda automatically
// and doesn't impact much given that it's just one kernel call.
__global__ void render(camera cam, vec3* framebuffer, hittable** world, hiprandState *rand_state) {
    cam.render(framebuffer, world, rand_state);
}

void save_framebuffer_to_file(vec3* framebuffer, int res_x, int res_y, std::string output_filename) {
    std::ofstream file(output_filename);
    std::streambuf* coutBuffer = std::cout.rdbuf();
    std::cout.rdbuf(file.rdbuf());

    std::cout << "P3\n" << res_x << " " << res_y << "\n255\n";
    for (int j = 0; j < res_y; j++) {
        for (int i = 0; i < res_x; i++) {
            size_t pixel_index = j*res_x + i;
            vec3 rgb = framebuffer[pixel_index];

            write_color(std::cout, rgb);
        }
    }

    std::cout.rdbuf(coutBuffer);
    std::cout << "Finished" << std::endl;
}

camera create_camera() {
    camera cam;
    cam.aspect_ratio = 16.0f/9.0f;
    cam.res_x  = 400;
    cam.spp    = 25;
    cam.center = point3(-2, 2, 1);
    cam.lookat = point3(0, 0, -1);
    cam.up     = point3(0, 1, 0);
    cam.vfov   = 30;

    cam.defocus_angle  = 0.1f;
    cam.focal_distance = 10.0f;
    cam.initialize();
    return cam;
}

int main() {
    // Camera
    camera cam = create_camera();
    int res_y  = cam.get_res_y();

    // Framebuffer
    int num_pixels = cam.res_x * res_y;
    size_t framebuffer_size = num_pixels * sizeof(vec3);
    vec3 *framebuffer;

    checkCudaErrors(hipMallocManaged((void **)&framebuffer, framebuffer_size));

    // materials: 
    material_manager* mat_manager;
    checkCudaErrors(hipMallocManaged((void**)&mat_manager, sizeof(mat_manager)));
    create_materials(mat_manager);
    mat_manager->dispatch();

    int threads_x = 8, threads_y = 8;
    dim3 threads(threads_x, threads_y);
    dim3 blocks(cam.res_x / threads_x  + 1, res_y / threads_y + 1);

    hiprandState *rand_state;
    checkCudaErrors(hipMalloc((void **)&rand_state, num_pixels * sizeof(hiprandState)));

    render_init<<<blocks, threads>>>(1984, cam.res_x, res_y, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    create_device_materials<<<1, 1>>>(
        mat_manager->get_device_material_info(),
        mat_manager->get_device_materials(),
        mat_manager->size()
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // world creation
    hittable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 22*22*sizeof(hittable *)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));

    create_world<<<1,1>>>(d_list,d_world, mat_manager, rand_state);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    rtweekend::clock c;

    c.start();
    render<<<blocks, threads>>>(cam, framebuffer, d_world, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    c.end();
    c.print();

    std::cout << "Saving framebuffer to file..." << std::endl;
    save_framebuffer_to_file(framebuffer, cam.res_x, res_y, "output/output.ppm");

    // freeing stuff
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(framebuffer));
    return 0;
}


